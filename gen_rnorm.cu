#include "hip/hip_runtime.h"
#include "CUDA_Rmath/Rmath.h"
#include <hiprand/hiprand_kernel.h>

__global__ void gen_rnorm_kernel(unsigned seed, double rbetas[5][7][10000][5], double *rnorms) {
	int id = blockDim.x*blockIdx.x+threadIdx.x;
	int i = id/10000, j = id;
	if (i < 70) i = 0;
	else if (i < 175) i = 1, j -= 700000;
	else if (i < 315) i = 2, j -= 1750000;
	else if (i < 490) i = 3, j -= 3150000;
	else if (i < 700) i = 4, j -= 4900000;
	int k = j%((i+2)*5*10000)/((i+2)*5);
	j /= ((i+2)*5*10000);
	
	hiprandState_t state;
	hiprand_init(seed, id, 0, &state);
	for (int l = 0; l < 5; l++) {
		rnorms[id*5+l] = rnorm(hiprand(&state), hiprand(&state), rbetas[i][j][k][l], 1);
	}
}
extern "C" void gen_rnorm(double *rbetas, double *rnorms)  {
	double *d_rbetas, *d_rnorms;
	hipMalloc(&d_rbetas, sizeof(double)*1750000);
	hipMalloc(&d_rnorms, sizeof(double)*35000000);
	hipMemcpy(d_rbetas, rbetas, sizeof(double)*1750000, hipMemcpyHostToDevice);
	
	gen_rnorm_kernel<<<7000, 1000>>>(time(0), (double (*)[7][10000][5])d_rbetas, d_rnorms);
	
	hipMemcpy(rnorms, d_rnorms, sizeof(double)*35000000, hipMemcpyDeviceToHost);
	hipFree(d_rbetas);
	hipFree(d_rnorms);
}