#include "hip/hip_runtime.h"
#include "Rmath.h"

__device__ double fmax2(double x, double y)
{
	if (isnan(x) || isnan(y))
		return x + y;
	return (x < y) ? y : x;
}