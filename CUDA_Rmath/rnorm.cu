#include "hip/hip_runtime.h"
#include "Rmath.h"

__device__ double rnorm(unsigned I1, unsigned I2, double mu, double sigma)
{
    if (isnan(mu) || !isfinite(sigma) || sigma < 0.)
	return NAN;
    if (sigma == 0. || !isfinite(mu))
	return mu; /* includes mu = +/- Inf with finite sigma */
    else
	return mu + sigma * norm_rand(I1, I2);
}