#include "hip/hip_runtime.h"
#include "Rmath.h"

__device__ double fmin2(double x, double y)
{
	if (isnan(x) || isnan(y))
		return x + y;
	return (x < y) ? x : y;
}