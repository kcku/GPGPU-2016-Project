#include "hip/hip_runtime.h"
#include "Rmath.h"

#define expmax (DBL_MAX_EXP * M_LN2)/* = log(DBL_MAX) */

__device__ double rbeta(unsigned I1, unsigned I2, double aa, double bb)
{
	double ur = unif_rand(I1, I2);
    double a, b, alpha;
    double r, s, t, u1, u2, v, w, y, z;

    int qsame;
    /* FIXME:  Keep Globals (properly) for threading */
    /* Uses these GLOBALS to save time when many rv's are generated : */
    double beta, gamma, delta, k1, k2;
    double olda = -1.0;
    double oldb = -1.0;

    if (aa <= 0. || bb <= 0. || (!isfinite(aa) && !isfinite(bb)))
	return NAN;

    if (!isfinite(aa))
    	return 1.0;

    if (!isfinite(bb))
    	return 0.0;

    /* Test if we need new "initializing" */
    qsame = (olda == aa) && (oldb == bb);
    if (!qsame) { olda = aa; oldb = bb; }

    a = fmin2(aa, bb);
    b = fmax2(aa, bb); /* a <= b */
    alpha = a + b;

#define v_w_from__u1_bet(AA) 			\
	    v = beta * log(u1 / (1.0 - u1));	\
	    if (v <= expmax) {			\
		w = AA * exp(v);		\
		if(!isfinite(w)) w = DBL_MAX;	\
	    } else				\
		w = DBL_MAX


    if (a <= 1.0) {	/* --- Algorithm BC --- */

	/* changed notation, now also a <= b (was reversed) */

	if (!qsame) { /* initialize */
	    beta = 1.0 / a;
	    delta = 1.0 + b - a;
	    k1 = delta * (0.0138889 + 0.0416667 * a) / (b * beta - 0.777778);
	    k2 = 0.25 + (0.5 + 0.25 / delta) * a;
	}
	/* FIXME: "do { } while()", but not trivially because of "continue"s:*/
	for(;;) {
	    u1 = ur;
	    u2 = ur;
	    if (u1 < 0.5) {
		y = u1 * u2;
		z = u1 * y;
		if (0.25 * u2 + z - y >= k1)
		    continue;
	    } else {
		z = u1 * u1 * u2;
		if (z <= 0.25) {
		    v_w_from__u1_bet(b);
		    break;
		}
		if (z >= k2)
		    continue;
	    }

	    v_w_from__u1_bet(b);

	    if (alpha * (log(alpha / (a + w)) + v) - 1.3862944 >= log(z))
		break;
	}
	return (aa == a) ? a / (a + w) : w / (a + w);

    }
    else {		/* Algorithm BB */

	if (!qsame) { /* initialize */
	    beta = sqrt((alpha - 2.0) / (2.0 * a * b - alpha));
	    gamma = a + 1.0 / beta;
	}
	do {
	    u1 = ur;
	    u2 = ur;

	    v_w_from__u1_bet(a);

	    z = u1 * u1 * u2;
	    r = gamma * v - 1.3862944;
	    s = a + r - w;
	    if (s + 2.609438 >= 5.0 * z)
		break;
	    t = log(z);
	    if (s > t)
		break;
	}
	while (r + alpha * log(alpha / (b + w)) < t);

	return (aa != a) ? b / (b + w) : w / (b + w);
    }
}